#include "hip/hip_runtime.h"
﻿/// 2015110758 류영석 20210409

#include "hip/hip_runtime.h"
#include ""

#include "..\usr\include\GL\freeglut.h";
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define FILE_NAME "../image2.raw"

const int Size = 512;
unsigned char* pSrcImage = NULL;
unsigned char* pOutImage = NULL;
bool flag = false;

void Render();
void Reshape(int w, int h);
void Keyboard(unsigned char key, int x, int y);
void SobelEdge();

#pragma region CUDA 함수변수들

#define TILE_WIDTH 1024

__global__ void SobelEdgeKernel(float* cpSobelResult, unsigned char* pSrcImage);
__global__ void SobelApplyKernel(float* pSobelResult, unsigned char* pOutImage, int min, int max);
__global__ void GetMaxKernel(float* pSobelResult, float* arrMinMax);

#pragma endregion

int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB);

	glutInitWindowSize(Size, Size);
	glutCreateWindow("Sobel Edge Detector(GPU)");

	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutKeyboardFunc(Keyboard);

	pSrcImage = new unsigned char[Size * Size];
	pOutImage = new unsigned char[Size * Size];

	FILE* infile;
	fopen_s(&infile, FILE_NAME, "rb");
	fread(pOutImage, sizeof(unsigned char), Size * Size, infile);
	for (int i = 0; i < Size * Size; ++i) {
		pSrcImage[i] = pOutImage[Size * Size - i - 1];
	}
	fclose(infile);

	clock_t st = clock();
	SobelEdge();
	printf("Elapsed time = %u ms\n", clock() - st);

	glutMainLoop();

	delete[] pSrcImage;
	delete[] pOutImage;
	return 0;
}

/// <summary>
/// Sobel 관련 기본 함수. 모든 CUDA 연산은 이 함수 안에서 창 띄우기 전에 끝나도록
/// </summary>
void SobelEdge()
{
	auto memorySizeFloat = sizeof(float) * Size * Size;
	auto memorySizeUChar = Size * Size * sizeof(unsigned char);

	// GPU 사용 설정
	hipSetDevice(0);

	// 소벨 결과를 담을 호스트 변수
	float* pSobelResult = new float[Size * Size];
	memset(pSobelResult, 0, memorySizeFloat);

	// pSobelResult의 값을 복사하여 디바이스에서 사용될 변수
	float* cpSobelResult;
	hipMalloc((void**)&cpSobelResult, memorySizeFloat);
	hipMemcpy(cpSobelResult, pSobelResult, memorySizeFloat, hipMemcpyHostToDevice);

	// pSrcImage또는 pOutImage를 복사하여 디바이스에서 사용될 변수
	unsigned char* cpyImage;
	hipMalloc((void**)&cpyImage, memorySizeUChar);
	hipMemcpy(cpyImage, pSrcImage, memorySizeUChar, hipMemcpyHostToDevice);

	// 여러 차례 계산해보았으나, 다차원 블록이 속도가 최소 10 ~ 최대 60ms까지 차이가 나서 1차원/1차원 방법을 사용했습니다.
	// 주석처리된 다차원 블록 계산시 실행시간 최저 133 최대 190, 현재 방법 최저 120 최대 144
	/*auto tWidth = (Size - 1) / TILE_WIDTH + 1;
	dim3 gridDim(tWidth, tWidth);
	dim3 blockDim(TILE_WIDTH, TILE_WIDTH);*/
	dim3 gridDim(Size, 1);
	dim3 blockDim(Size, 1);

	// 소벨 계산 커널함수 실행 후 완료 대기. 이후 호스트 변수로 결과 복사
	SobelEdgeKernel << <gridDim, blockDim, 1 >> > (cpSobelResult, cpyImage);
	hipDeviceSynchronize();
	hipMemcpy(pSobelResult, cpSobelResult, memorySizeFloat, hipMemcpyDeviceToHost);

	// 일반적인 방법으로 커널 함수로 바꿀 시 필터 결과에 문제 생김. min max 변수 공유 문제로 보임
	float min = 10000000.0f, max = -10000000.0f;
	for (int i = 1; i < Size - 1; i++) {
		for (int j = 1; j < Size - 1; j++) {
			int idx = i * Size + j;
			min = (pSobelResult[idx] < min) ? pSobelResult[idx] : min;
			max = (pSobelResult[idx] > max) ? pSobelResult[idx] : max;
		}
	}

	// cpyImage 재사용
	hipFree(cpyImage);
	hipMalloc((void**)&cpyImage, memorySizeUChar);
	hipMemcpy(cpyImage, pOutImage, memorySizeUChar, hipMemcpyHostToDevice);

	// 소벨 적용 커널함수 실행 후 완료 대기. 이후 호스트 변수로 결과 복사
	SobelApplyKernel << < gridDim, blockDim, 1 >> > (cpSobelResult, cpyImage, min, max);
	hipDeviceSynchronize();
	hipMemcpy(pOutImage, cpyImage, memorySizeUChar, hipMemcpyDeviceToHost);

	// 전체 변수들 메모리 해제 및 CUDA 계산 종료.
	delete[] pSobelResult;
	hipFree(cpSobelResult);
	hipFree(cpyImage);
	hipDeviceReset();
}

void Render()
{
	glClearColor(1.0, 1.0, 1.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);

	if (flag == true) {
		glDrawPixels(Size, Size, GL_LUMINANCE, GL_UNSIGNED_BYTE, pOutImage);
	}
	else {
		glDrawPixels(Size, Size, GL_LUMINANCE, GL_UNSIGNED_BYTE, pSrcImage);
	}

	glFinish();
}

void Reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void Keyboard(unsigned char key, int x, int y)
{
	// 27=esc
	if (key == 27) {
		//glutLeaveMainLoop();
		exit(-1);
	}

	if (key == 's') {
		flag = !flag;
	}

	glutPostRedisplay();
}

#pragma region kernel functions
/// <summary>
/// 소벨 엣지 검출 커널 함수. 3중 루프문에서 바깥 2중 루프문을 간략화.  
/// </summary>
/// <param name="cpSobelResult"></param>
/// <param name="pSrcImage"></param>
/// <returns></returns>
__global__ void SobelEdgeKernel(float* cpSobelResult, unsigned char* pSrcImage)
{
	int MaskSobelX[] = {
		-1, 0, 1,
		-2,0,2,
		-1,0,1
	};

	int MaskSobelY[] = {
		1,2, 1,
		0,0,0,
		-1,-2,-1
	};

	int i = blockIdx.x;
	int j = threadIdx.x;
	//int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
	//int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

	// 지워서 원본 for문보다 아래위로 1씩 더 보게 되면 끄트머리 1픽셀 백화
	if (i <= 0 || j <= 0 || i >= Size - 1 || j >= Size - 1) {
		return;
	}

	int Gx = 0, Gy = 0;
	// 누적하는 값이므로 변수 공유 대책 없이는 커널화 하지 않음
	for (int k = 0; k < 9; ++k) {
		int r = k / 3, c = k % 3;
		int idx = (i + r - 1) * Size + j + c - 1;
		Gx = Gx + MaskSobelX[k] * pSrcImage[idx];
		Gy = Gy + MaskSobelY[k] * pSrcImage[idx];
	}

	cpSobelResult[i * Size + j] = sqrtf(Gx * Gx + Gy * Gy);
}

/// <summary>
/// 소벨 필터 값에 따라 이미지에 적용하는 커널 함수.
/// </summary>
/// <param name="pSobelResult"></param>
/// <param name="pOutImage"></param>
/// <param name="min"></param>
/// <param name="max"></param>
/// <returns></returns>
__global__ void SobelApplyKernel(float* pSobelResult, unsigned char* pOutImage, int min, int max) {
	int i = blockIdx.x;
	int j = threadIdx.x;

	//int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
	//int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

	if (i <= 1 || j <= 1 || i >= Size - 1 || j >= Size) {
		return;
	}
	int idx = i * Size + j;
	float t = (pSobelResult[idx] - min) / (max - min);
	pOutImage[idx] = (unsigned char)(255 * t);
}

/// <summary>
/// SobelEdge 함수에서 min max 값 구할 때 사용하려 했던 함수. 변수 공유 관련 적용 안했으므로 미사용
/// </summary>
/// <param name="pSobelResult"></param>
/// <param name="arrMinMax"></param>
/// <returns></returns>
__global__ void GetMaxKernel(float* pSobelResult, float* arrMinMax) {

	int i = blockIdx.x;
	int j = threadIdx.x;

	/*int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int j = blockIdx.x * TILE_WIDTH + threadIdx.x;*/

	if (i <= 1 || j <= 1 || i >= Size - 1 || j >= Size - 1) {
		return;
	}

	int idx = i * Size + j;

	arrMinMax[0] = (pSobelResult[idx] < arrMinMax[0]) ? pSobelResult[idx] : arrMinMax[0];
	arrMinMax[1] = (pSobelResult[idx] > arrMinMax[1]) ? pSobelResult[idx] : arrMinMax[1];
}
#pragma endregion
